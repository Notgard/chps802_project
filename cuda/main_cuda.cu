#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "config.h"

#define THREADS_PER_BLOCK 16

/// @brief Stores the linear system read from the given file into a linear system structure
/// @param input_filename the given file's path
/// @param linear_system the linear system to create from the given file
void cuda_read_linear_system_from_file(char *input_filename, linear_system_t *linear_system)
{
    FILE *file;

    int i;
    int status;
    int linear_system_unknowns, nb_matrix_rows, nb_matrix_cols;
    double file_value_pointer;

    // open the file in read-only mode
    if (!(file = fopen(input_filename, "r")))
    {
        fprintf(stderr, "Error reading file %s!\n", input_filename);
        exit(EXIT_FAILURE);
    }

    // read linear system configuration from given file
    if ((status = fscanf(file, "%d", &linear_system_unknowns)) == EOF)
    {
        fprintf(stderr, "[%d]Reached End Of File\n", status);
        exit(EXIT_FAILURE);
    }

    nb_matrix_rows = linear_system_unknowns;
    nb_matrix_cols = nb_matrix_rows + 1;

    int total_values = (nb_matrix_rows) * (nb_matrix_cols);

    // allocate memory for storing structures and check for out of memory allocations
    double *linear_system_matrix = (double *)malloc(sizeof(double) * total_values);
    if (linear_system_matrix == NULL)
    {
        fprintf(stderr, "Out of memory!\n");
        exit(EXIT_FAILURE);
    }

    double **pointer_storage_array = (double **)malloc(sizeof(double *) * nb_matrix_rows);
    if (pointer_storage_array == NULL)
    {
        fprintf(stderr, "Out of memory!\n");
        exit(EXIT_FAILURE);
    }

    // store the pointers into the pointer storage array
    for (i = 0; i < nb_matrix_rows; i++)
    {
        pointer_storage_array[i] = &linear_system_matrix[i * nb_matrix_cols];
    }

    double **vector_b_storage = (double **)malloc(sizeof(double *) * nb_matrix_rows);
    if (vector_b_storage == NULL)
    {
        fprintf(stderr, "Out of memory!\n");
        exit(EXIT_FAILURE);
    }

    // store the pointers into the pointer storage array
    for (i = 0; i < nb_matrix_rows; i++)
    {
        vector_b_storage[i] = &linear_system_matrix[i * nb_matrix_cols + nb_matrix_cols - 1];
    }

    double ***matrix_a = (double ***)malloc(sizeof(double **) * nb_matrix_rows);
    if (matrix_a == NULL)
    {
        fprintf(stderr, "Out of memory!\n");
        exit(EXIT_FAILURE);
    }

    // store the pointers into the pointer storage array
    for (i = 0; i < nb_matrix_rows; i++)
    {
        matrix_a[i] = (double **)malloc(sizeof(double *) * nb_matrix_rows);
    }

    // read the values of the linear system from the file and write them to the matrix
    for (i = 0; i < total_values; i++)
    {
        // read values from file as double
        if ((status = fscanf(file, "%lf", &file_value_pointer)) == EOF)
        {
            fprintf(stderr, "Reached End Of File\n");
            exit(EXIT_FAILURE);
        }

        linear_system_matrix[i] = file_value_pointer;
    }

    // copy content of linear system matrix to matrix_a
    for (i = 0; i < nb_matrix_rows; i++)
    {
        for (int j = 0; j < nb_matrix_rows; j++)
        {
            matrix_a[i][j] = &linear_system_matrix[i * nb_matrix_cols + j];
        }
    }

    // insert the extracted linear system matrix into the structure
    linear_system->data = linear_system_matrix; // matrice augmentée
    linear_system->storage = pointer_storage_array;

    linear_system->nb_unknowns = linear_system_unknowns;
    linear_system->vec_B = vector_b_storage;
    linear_system->matrix_A = matrix_a;

    if ((status = fclose(file)) == EOF)
    {
        fprintf(stderr, "Can't close file %s\n", input_filename);
        exit(EXIT_FAILURE);
    }
}

/// @brief Writes the result of the solved linear system into a file
/// @param output_filename the ouput file's path
/// @param linear_system the linear system to write into the given output_file
/// @param solutions the solutions for the given linear system
void cuda_write_linear_system_to_file(char *output_filename, linear_system_t *linear_system, double *solutions)
{
    FILE *file;

    int i;
    int status;

    int nb_matrix_rows = linear_system->nb_unknowns;

    // open the file in write mode, creates the file if doesn't exist
    if (!(file = fopen(output_filename, "w+")))
    {
        fprintf(stderr, "Error opening file %s\n", output_filename);
        exit(EXIT_FAILURE);
    }

    // write the number of unknowns in the linear system
    if ((status = fprintf(file, "%d\n", linear_system->nb_unknowns)) == EOF)
    {
        fprintf(stderr, "status: %d\n", status);
        perror("Can't write content to output file");
        exit(EXIT_FAILURE);
    }

    // loop over the 1D linear system matrix using the storage pointers

/*     int nb_matrix_cols = nb_matrix_rows + 1;
    for (i = 0; i < nb_matrix_rows; i++)
    {
        for (int j = 0; j < nb_matrix_cols; j++)
        {
            if (linear_system->storage[i][j] != 0.0f)
            {
                // write the contents of the linear system matrix into the output file
                if ((status = fprintf(file, "%.3lf ", linear_system->storage[i][j])) == EOF)
                {
                    perror("Can't write linear system to output file");
                    exit(EXIT_FAILURE);
                }
            }
            else if (linear_system->storage[i][j] != 0.0f && (linear_system->storage[i][j] < 1 && linear_system->storage[i][j] > -1))
            {                    // truncate the floating point value leading zero
                char buffer[20]; // Assuming a maximum length of the printed number
                double truncate_value = linear_system->storage[i][j];
                sprintf(buffer, "%.3lf", truncate_value);

                if ((status = fprintf(file, "%s ", buffer + 1)) == EOF)
                {
                    perror("Can't write linear system to output file");
                    exit(EXIT_FAILURE);
                }
            }
        }
        // add linebreak to file for each written row from the linear system matrix
        if ((status = fprintf(file, "%s", "\n")) == EOF)
        {
            perror("Can't write linebreak to output file");
            exit(EXIT_FAILURE);
        }
    } */

    if (solutions != NULL)
    {
        for (i = 0; i < nb_matrix_rows; i++)
        {
            // write solutions to the linear system to the file (if they exist)
            if ((status = fprintf(file, "%.3lf ", solutions[i])) == EOF)
            {
                perror("Can't write linear system to output file");
                exit(EXIT_FAILURE);
            }
        }
    }

    if ((status = fclose(file)) == EOF)
    {
        fprintf(stderr, "Can't close file %s\n", output_filename);
        exit(EXIT_FAILURE);
    }
}

/// @brief frees the memory from the linear system structure members
/// @param linear_system
void cuda_clean_linear_system_memory(linear_system_t *linear_system)
{
    free(linear_system->storage);
    free(linear_system->data);
    free(linear_system->vec_B);
    free(linear_system->matrix_A);
}

void cuda_print_linear_system_matrix(linear_system_t *linear_system)
{
    int i, j;

    int nb_matrix_rows = linear_system->nb_unknowns;
    int nb_matrix_cols = nb_matrix_rows + 1;

    for (i = 0; i < nb_matrix_rows; i++)
    {
        for (j = 0; j < nb_matrix_cols; j++)
        {
            printf("%-10.3lf ", linear_system->storage[i][j]);
        }
        printf("\n");
    }
}

double *cuda_solve_linear_system(linear_system_t *linear_system)
{
    int j = 0;
    double result = 0;

    int nb_matrix_rows = linear_system->nb_unknowns;
    int nb_matrix_cols = nb_matrix_rows + 1;

    // allocate memory for the solutions array, initialized with zeros
    double *solutions = (double *)calloc(linear_system->nb_unknowns, sizeof(double));
    if (solutions == NULL)
    {
        fprintf(stderr, "Out of memory!\n");
        exit(EXIT_FAILURE);
    }

    for (int i = nb_matrix_rows - 1; i >= 0; i--)
    { // commencer à la dernière ligne
        result = 0;
        for (j = i; j < nb_matrix_cols - 1; j++)
        {
            if (i != j)
            { // if the the two aren't the samen then an initial solution has been found
                // result += A[i][j] * R[j]
                result += linear_system->storage[i][j] * solutions[j];
            }
        }
        // R[i]=(A[i][dim-1]-result)/A[i][i]
        solutions[i] = (linear_system->storage[i][nb_matrix_cols - 1] - result) / linear_system->storage[i][i];
    }
    return solutions;
}

void cuda_swap_linear_system_rows(linear_system_t *linear_system, int row1, int row2)
{
    int i;
    int nb_matrix_cols = linear_system->nb_unknowns + 1;

    double temp_value;

    for (i = 0; i < nb_matrix_cols; i++)
    {
        temp_value = linear_system->storage[row1][i];
        linear_system->storage[row1][i] = linear_system->storage[row2][i];
        linear_system->storage[row2][i] = temp_value;
    }
}

// gauss elimination function taking the linear system matrix and loading the pivot line into shared memory
__global__ void find_pivot_and_swap(double *d_linear_system, int n_rows, int n_cols, int current_line, int *d_pivot_line)
{
    extern __shared__ double shared_data[];

    int tid = threadIdx.x;
    int col = current_line;
    double *local_max = shared_data;
    int *local_pivot = (int *)&local_max[blockDim.x];

    // Step 1: Identify the pivot row
    double pivot = 0;
    double p = d_linear_system[current_line * n_cols + col];
    double abs_val;
    double col_val;
    double p_abs;
    int pivot_row = current_line;
    for (int row = tid + current_line + 1; row < n_rows; row += blockDim.x)
    {
        col_val = d_linear_system[row * n_cols + col];
        abs_val = fabs(col_val); // check if the absolute value of the pivot coefficient to be selected
        p_abs = fabs(p);
        pivot = MAX(p_abs, abs_val);
        if (p_abs != pivot)
            pivot_row = row;
        p = (pivot == abs_val) ? col_val : p;
    }

    __syncthreads();

    local_max[tid] = p;
    local_pivot[tid] = pivot_row;

    __syncthreads();

    // Reduce to find the maximum value and corresponding row index
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            if (fabs(local_max[tid]) < fabs(local_max[tid + stride]))
            {
                local_max[tid] = local_max[tid + stride];
                local_pivot[tid] = local_pivot[tid + stride];
            }
        }
        __syncthreads();
    }

    __syncthreads();

    if(tid == 0) {
        *d_pivot_line = local_pivot[0];
    }
/*     if (tid == 0)
    {
        if (current_line != local_pivot[0])
        {
            // Swap rows current_line and pivot_row
            for (int j = 0; j < n_cols; j++)
            {
                double temp = d_linear_system[current_line * n_cols + j];
                d_linear_system[current_line * n_cols + j] = d_linear_system[local_pivot[0] * n_cols + j];
                d_linear_system[local_pivot[0] * n_cols + j] = temp;
            }
        }
    }
    __syncthreads(); */
}

// paralell selection of the max absolute value gaussian pivot
template <int BLOCK_SIZE>
__global__ void gauss_elimination(double *d_linear_system, int n_rows, int n_cols, int pivot_line)
{
    // Thread ID
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // Allocate shared memory for the pivot row
    extern __shared__ double pivot_row[];

    if (row < n_rows && col < n_cols)
    {
        // Load the pivot row into shared memory
        if (row == pivot_line)
        {
            pivot_row[col] = d_linear_system[pivot_line * n_cols + col];
        }
        __syncthreads();

        // Perform Gaussian elimination for elements in rows below the pivot line
        if (row > pivot_line)
        {
            double pivot_value = pivot_row[pivot_line];
            double factor = d_linear_system[row * n_cols + pivot_line] / pivot_value;

            if (col >= pivot_line)
            {
                d_linear_system[row * n_cols + col] -= factor * d_linear_system[pivot_line * n_cols + col];
            }
        }
        __syncthreads();
    }
}

double GBPerSec(int bytes, double sec)
{
    return (double)(bytes) / (1024. * 1024. * 1024.) / sec;
}

void cudaMallocCheckError(void **addr, size_t size)
{
    hipError_t code;
    code = hipMalloc(addr, size);
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(code));
        exit(EXIT_FAILURE);
    }
}

void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}

void pivot_de_gauss(linear_system_t *h_linear_system)
{
    double *d_linear_system = NULL;
    int* d_pivot_line = NULL;

    hipError_t code;
    clock_t start, end;
    double elapsed;
    int n = h_linear_system->nb_unknowns;
    int h_system_size = n * (n + 1);
    size_t h_size = (n * (n + 1)) * sizeof(double);

    int h_pivot_line;

    cudaMallocCheckError((void **)&d_linear_system, h_size);
    cudaMallocCheckError((void **)&d_pivot_line, sizeof(int));

    code = hipMemcpy(d_linear_system, h_linear_system->data, h_size, hipMemcpyHostToDevice);

    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(code));
        exit(EXIT_FAILURE);
    }

    const int nb_blocks = (h_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    dim3 blockSize(32, 32);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, ((n + 1) + blockSize.y - 1) / blockSize.y);
    int sharedMemSize = (n + 1) * sizeof(double);

    start = clock();
    for (int curr_line = 0; curr_line < n; curr_line++)
    {
        int pivot_line = curr_line;

        find_pivot_and_swap<<<nb_blocks, THREADS_PER_BLOCK, sharedMemSize>>>(d_linear_system, n, n + 1, pivot_line, d_pivot_line);
        code = hipGetLastError();
        if (code != hipSuccess)
        {
            fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(code));
            exit(EXIT_FAILURE);
        }
        hipDeviceSynchronize();

        hipMemcpy(&h_pivot_line, d_pivot_line, sizeof(int), hipMemcpyDeviceToHost);
        //printf("Pivot line %d: %d\n", curr_line, h_pivot_line);

        hipMemcpy(h_linear_system->data, d_linear_system, h_size, hipMemcpyDeviceToHost);
        if(pivot_line != h_pivot_line) {
            cuda_swap_linear_system_rows(h_linear_system, pivot_line, h_pivot_line);
        }

/*         printf("After pivot line %d:\n", curr_line);
        // print the h_linear_system->data
        for (int i = 0; i < n; i++)
        {
            for (int j = 0; j < n + 1; j++)
            {
                printf("%-10.3lf ", h_linear_system->data[i * (n + 1) + j]);
            }
            printf("\n");
        }
        printf("---------------------------------------\n"); */

        hipMemcpy(d_linear_system, h_linear_system->data, h_size, hipMemcpyHostToDevice);
        
        gauss_elimination<1024><<<gridSize, blockSize, sharedMemSize>>>(d_linear_system, n, n + 1, pivot_line);
        hipDeviceSynchronize();
/*         hipMemcpy(h_linear_system->data, d_linear_system, h_size, hipMemcpyDeviceToHost);
        // print the h_linear_system->data
        printf("After gauss elimination\n");
        for (int i = 0; i < n; i++)
        {
            for (int j = 0; j < n + 1; j++)
            {
                printf("%-10.3lf ", h_linear_system->data[i * (n + 1) + j]);
            }
            printf("\n");
        }
        printf("---------------------------------------\n"); */
        code = hipGetLastError();
        if (code != hipSuccess)
        {
            fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(code));
            exit(EXIT_FAILURE);
        }
    }
    end = clock();
    elapsed = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Gaussian elimination time in CUDA: %.5lf\n", elapsed);

    hipMemcpy(h_linear_system->data, d_linear_system, h_size, hipMemcpyDeviceToHost);
    printf("Final linear system: \n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n + 1; j++)
        {
            printf("%f ", h_linear_system->data[i * (n + 1) + j]);
        }
        printf("\n");
    }
}

#define OUT_FILE "output.txt"

int main(int argc, char *argv[])
{
    printf("Starting %s...\n", argv[0]);

    if (argc < 2)
    {
        fprintf(stderr, "Incorrect arguments : No given filename!\n");
        exit(EXIT_FAILURE);
    }

    char *filename = argv[1];
    clock_t start, end;
    double elapsed;

    printCudaInfo();

    linear_system_t linear_system = {.nb_unknowns = 0, .data = NULL, .storage = NULL};

    cuda_read_linear_system_from_file(filename, &linear_system);

    start = clock();
    pivot_de_gauss(&linear_system);
    end = clock();

    printf("\n------------------------------------------\n");

    cuda_print_linear_system_matrix(&linear_system);

    double *solutions = cuda_solve_linear_system(&linear_system);

    cuda_write_linear_system_to_file(OUT_FILE, &linear_system, solutions);

    cuda_clean_linear_system_memory(&linear_system);

    free(solutions);

    elapsed = (double)(end - start) / CLOCKS_PER_SEC;
    double GBs = GBPerSec(linear_system.nb_unknowns * (linear_system.nb_unknowns + 1) * sizeof(double), elapsed);
    printf("Overall execution time for %f Gigabytes per second : %.5lf\n", GBs, elapsed);
    printf("----------------------------------------------------------------------------------\n");

    return EXIT_SUCCESS;
}